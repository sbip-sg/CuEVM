#include "hip/hip_runtime.h"
#include <CuEVM/utils/python_utils.h>
#include <Python.h>
#include <getopt.h>

#include <CuEVM/utils/evm_utils.cuh>
#include <chrono>
#include <fstream>

using namespace python_utils;
// define the kernel function
__global__ void kernel_evm_multiple_instances(cgbn_error_report_t* report, CuEVM::evm_instance_t* instances,
                                              uint32_t count) {
    int32_t instance = (blockIdx.x * blockDim.x + threadIdx.x) / CuEVM::cgbn_tpi;
    if (instance >= count) return;
    CuEVM::ArithEnv arith(cgbn_no_checks, report, instance);
    // CuEVM::bn_t test;
    // printf("print simplified trace data device\n");
    // instances[instance].simplified_trace_data_ptr->print();
// printf("new instance %d\n", instance);
#ifdef EIP_3155
    __ONE_GPU_THREAD_WOSYNC_BEGIN__
    printf("instance %d\n", instance);
    printf("world state\n");
    instances[instance].world_state_data_ptr->print();
    printf("touch state\n");
    instances[instance].touch_state_data_ptr->print();
    printf("instance %d\n", instance);
    printf("transaction\n");
    instances[instance].transaction_ptr->print();
    __ONE_GPU_THREAD_WOSYNC_END__
#endif
    __SHARED_MEMORY__ CuEVM::evm_message_call_t shared_message_call;
    __SHARED_MEMORY__ CuEVM::evm_word_t shared_stack[CuEVM::shared_stack_size];
    CuEVM::evm_t* evm = new CuEVM::evm_t(arith, instances[instance], &shared_message_call, shared_stack);
    CuEVM::cached_evm_call_state cached_state(arith, evm->call_state_ptr);
    // printf("\nevm->run(arith) instance %d\n", instance);
    // printf("print simplified trace data device inside evm\n");
    // evm->simplified_trace_data_ptr->print();
    __SYNC_THREADS__
    evm->run(arith, cached_state);

#ifdef EIP_3155
    if (instance == 0) {
        __ONE_GPU_THREAD_BEGIN__
        // instances[0].tracer_ptr->print(arith);
        instances[0].tracer_ptr->print_err();
        __ONE_GPU_THREAD_WOSYNC_END__
    }
#endif
    // print the final world state
    // __ONE_GPU_THREAD_WOSYNC_BEGIN__
    // instances[instance].world_state_data_ptr->print();
    // printf("simplified trace data\n");
    // instances[instance].simplified_trace_data_ptr->print();
    // __ONE_GPU_THREAD_WOSYNC_END__
    // delete evm;
    // evm = nullptr;
}

PyObject* run_interpreter_pyobject(PyObject* read_roots) {
    CuEVM::evm_instance_t* instances_data;
    CuEVM::ArithEnv arith(cgbn_no_checks, 0);
    printf("Running the interpreter\n");
#ifndef GPU
    printf("CPU libcuevm is not supported at the moment\n");
    return NULL;
#endif
    CUDA_CHECK(hipSetDevice(0));
    CUDA_CHECK(hipDeviceReset());
    printf("Running on GPU\n");
    cgbn_error_report_t* report;
    CUDA_CHECK(cgbn_error_report_alloc(&report));
    hipEvent_t start, stop;
    float milliseconds = 0;

    size_t size_value;
    hipDeviceGetLimit(&size_value, hipLimitStackSize);
    printf("current stack size %zu\n", size_value);
    hipDeviceGetLimit(&size_value, hipLimitStackSize);
    printf("current heap size %zu\n", size_value);
    size_t heap_size = (size_t(500) << 20);  // 500MB
    CUDA_CHECK(hipDeviceSetLimit(hipLimitMallocHeapSize, heap_size));
    CUDA_CHECK(hipDeviceSetLimit(hipLimitStackSize, 2 * 1024));
    hipDeviceGetLimit(&size_value, hipLimitStackSize);
    printf("current stack size %zu\n", size_value);
    CUDA_CHECK(hipDeviceSynchronize());
    // CUDA_CHECK(hipEventCreate(&start));
    // CUDA_CHECK(hipEventCreate(&stop));

    // read the json file with the global state

    uint32_t num_instances = 0;
    uint32_t managed = 1;

    if (!PyList_Check(read_roots)) {
        PyErr_SetString(PyExc_TypeError, "Argument must be a list of dictionaries.");
        return NULL;
    }

    Py_ssize_t count = PyList_Size(read_roots);
    // transaction_data_t* transactions = getTransactionDataFromPyObject(arith, PyDict_GetItemString(read_root,
    // "transaction"));
    // void get_evm_instances_from_PyObject(CuEVM::evm_instance_t * &evm_instances, const cJSON* test_json,
    //                                      uint32_t& num_instances);

    python_utils::get_evm_instances_from_PyObject(instances_data, read_roots, num_instances);
    // printf("print simplified trace data host\n");
    // instances_data[0].simplified_trace_data_ptr->print();
    printf("Running on GPU %d %d\n", num_instances, CuEVM::cgbn_tpi);
    // run the evm
    kernel_evm_multiple_instances<<<num_instances, CuEVM::cgbn_tpi>>>(report, instances_data, num_instances);
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipGetLastError());
    printf("GPU kernel finished\n");
    CGBN_CHECK(report);

    // printf("\n\ntesting world state printing on host\n\n");
    // instances_data[0].serialized_worldstate_data_ptr->print();
    // printf("print simplified trace data host\n");
    // for (uint32_t i = 0; i < num_instances; i++) {
    //     printf("\n\ninstance %d\n", i);
    //     instances_data[i].simplified_trace_data_ptr->print();
    // }
    PyObject* write_root = python_utils::pyobject_from_evm_instances(instances_data, num_instances);

    CuEVM::free_evm_instances(instances_data, num_instances, managed);

    CUDA_CHECK(cgbn_error_report_free(report));
    CUDA_CHECK(hipDeviceReset());
    return write_root;
    // Py_RETURN_NONE;
}

static PyObject* run_dict(PyObject* self, PyObject* args) {
    PyObject* read_root;

    // Parse the input PyObject* to get the Python object (dictionary)
    if (!PyArg_ParseTuple(args, "O", &read_root)) {
        return NULL;  // If parsing fails, return NULL
    }

    PyObject* write_root = run_interpreter_pyobject(read_root);
    // Return the resulting PyObject* (no need for manual memory management on Python side)
    return write_root;
}

static PyObject* print_dict(PyObject* self, PyObject* args) {
    PyObject* dict;

    // Parse the Python argument (a dictionary)
    if (!PyArg_ParseTuple(args, "O", &dict)) return nullptr;

    // Ensure the object is a dictionary
    if (!PyDict_Check(dict)) {
        PyErr_SetString(PyExc_TypeError, "Argument must be a dictionary.");
        return nullptr;
    }
    // Start recursive printing with no indent
    print_dict_recursive(dict, 0);

    Py_RETURN_NONE;
}

// Method definition
static PyMethodDef ExampleMethods[] = {{"print_dict", print_dict, METH_VARARGS, "Print dictionary keys and values."},
                                       {"run_dict", run_dict, METH_VARARGS, "Run the interpreter with a JSON object."},
                                       {nullptr, nullptr, 0, nullptr}};

// Module definition
static PyModuleDef examplemodule = {PyModuleDef_HEAD_INIT,
                                    "libcuevm",  // Module name
                                    nullptr,     // Module documentation
                                    -1,          // Size of per-interpreter state of the module
                                    ExampleMethods};

// Initialization function
PyMODINIT_FUNC PyInit_libcuevm(void) { return PyModule_Create(&examplemodule); }

// deprecated strings interfaces
/*
extern "C" char* run_json_string(const char* read_json_string) {
    cJSON *read_root = cJSON_Parse(read_json_string);
    if (read_root == NULL) {
        // Handle parsing error (optional)
        return NULL;
    }

    cJSON *write_root = cJSON_CreateObject();

    // Assume run_interpreter modifies write_root based on read_root
    run_interpreter(read_root, write_root);
    cJSON_Delete(read_root);
    char *json_str = cJSON_Print(write_root);
    cJSON_Delete(write_root);

    return json_str; // Caller needs to free this memory
}
*/
