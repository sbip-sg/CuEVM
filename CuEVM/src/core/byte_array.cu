#include "hip/hip_runtime.h"
// CuEVM: CUDA Ethereum Virtual Machine implementation
// Copyright 2023 Stefan-Dan Ciocirlan (SBIP - Singapore Blockchain Innovation
// Programme) Author: Stefan-Dan Ciocirlan Data: 2024-06-20
// SPDX-License-Identifier: MIT

#include <CuEVM/core/byte_array.cuh>
#include <CuEVM/utils/error_codes.cuh>
#include <CuEVM/utils/evm_utils.cuh>

namespace CuEVM {
__host__ __device__ byte_array_t::byte_array_t(const uint32_t size) : size(size) {
    __SHARED_MEMORY__ uint8_t *tmp_data[CGBN_IBP];
    // printf("byte_array_t::byte_array_t construction %d %d instance id %d\n", THREADIDX, THREAD_IDX_PER_INSTANCE,
    //        INSTANCE_IDX_PER_BLOCK);
    if (size > 0) {
        __ONE_GPU_THREAD_WOSYNC_BEGIN__
        tmp_data[INSTANCE_IDX_PER_BLOCK] = new uint8_t[size];
        __ONE_GPU_THREAD_END__
        // memset(tmp_data, 0, size * sizeof(uint8_t));
        // parallel_copy
        // printf("instance id %d , instance data %p\n", INSTANCE_IDX_PER_BLOCK, tmp_data[INSTANCE_IDX_PER_BLOCK]);
        uint32_t my_idx = THREAD_IDX_PER_INSTANCE;
        for (; my_idx < size; my_idx += CuEVM::cgbn_tpi) {
            tmp_data[INSTANCE_IDX_PER_BLOCK][my_idx] = 0;
        }
    } else
        tmp_data[INSTANCE_IDX_PER_BLOCK] = nullptr;
    __SYNC_THREADS__
    data = tmp_data[INSTANCE_IDX_PER_BLOCK];
}

__device__ byte_array_t::byte_array_t(uint8_t *data, uint32_t size) : size(size) {
    __SHARED_MEMORY__ uint8_t *tmp_data[CGBN_IBP];

    if (size > 0) {
        __ONE_GPU_THREAD_WOSYNC_BEGIN__
        tmp_data[INSTANCE_IDX_PER_BLOCK] = new uint8_t[size];
        __ONE_GPU_THREAD_END__
        // memcpy(tmp_data, data, size * sizeof(uint8_t));
        // parallel_copy
        uint32_t my_idx = THREAD_IDX_PER_INSTANCE;
        for (; my_idx < size; my_idx += CuEVM::cgbn_tpi) {
            tmp_data[INSTANCE_IDX_PER_BLOCK][my_idx] = data[my_idx];
        }
    } else
        tmp_data[INSTANCE_IDX_PER_BLOCK] = nullptr;

    this->data = tmp_data[INSTANCE_IDX_PER_BLOCK];
}

__host__ __device__ byte_array_t::byte_array_t(const byte_array_t &src_byte_array, uint32_t offset, uint32_t size)
    : size(size) {
    __SHARED_MEMORY__ uint8_t *tmp_data[CGBN_IBP];
    // printf("byte_array_t::byte_array_t %d %d %d %d\n", THREADIDX, THREAD_IDX_PER_INSTANCE, offset, size);
    if (size > 0) {
        __ONE_GPU_THREAD_WOSYNC_BEGIN__
        tmp_data[INSTANCE_IDX_PER_BLOCK] = new uint8_t[size];
        __ONE_GPU_THREAD_END__
        // memset(tmp_data, 0, size * sizeof(uint8_t));
        // parallel_copy
        uint32_t my_idx = THREAD_IDX_PER_INSTANCE;
        for (uint32_t idx = my_idx; idx < size; idx += CuEVM::cgbn_tpi) {
            tmp_data[INSTANCE_IDX_PER_BLOCK][idx] = 0;
        }
        // parallel_copy
        if (offset < src_byte_array.size)
            for (uint32_t idx = my_idx; idx < min(size, src_byte_array.size - offset); idx += CuEVM::cgbn_tpi) {
                tmp_data[INSTANCE_IDX_PER_BLOCK][idx] = src_byte_array.data[offset + idx];
            }
        // if (offset < src_byte_array.size)
        //     memcpy(tmp_data, src_byte_array.data + offset, min(size, src_byte_array.size - offset) *
        //     sizeof(uint8_t));
    } else
        tmp_data[INSTANCE_IDX_PER_BLOCK] = nullptr;
    // __ONE_GPU_THREAD_END__
    this->data = tmp_data[INSTANCE_IDX_PER_BLOCK];
}

__host__ byte_array_t::byte_array_t(const char *hex_string, int32_t endian, PaddingDirection padding)
    : size(0), data(nullptr) {
    from_hex(hex_string, endian, padding, 0);
}

__host__ byte_array_t::byte_array_t(const char *hex_string, uint32_t size, int32_t endian, PaddingDirection padding)
    : size(size), data(nullptr) {
    from_hex(hex_string, endian, padding, 0);
}

__host__ __device__ byte_array_t::~byte_array_t() { free(); }

__host__ __device__ void byte_array_t::free() {
    if ((size > 0) && (data != nullptr)) {
        // __ONE_GPU_THREAD_WOSYNC_BEGIN__
        // delete[] data;
        // __ONE_GPU_THREAD_WOSYNC_END__
        clear();
    }
}
__host__ void byte_array_t::free_managed() {
    if ((size > 0) && (data != nullptr)) {
        CUDA_CHECK(hipFree(data));
    }
    clear();
}
__host__ __device__ void byte_array_t::clear() {
    data = nullptr;
    size = 0;
}

__host__ __device__ byte_array_t::byte_array_t(const byte_array_t &other) : size(other.size) {
    __SHARED_MEMORY__ uint8_t *tmp_data[CGBN_IBP];
    __ONE_GPU_THREAD_BEGIN__
    if (size > 0) {
        tmp_data[INSTANCE_IDX_PER_BLOCK] = new uint8_t[size];
        memcpy(tmp_data[INSTANCE_IDX_PER_BLOCK], other.data, size * sizeof(uint8_t));
    } else
        tmp_data[INSTANCE_IDX_PER_BLOCK] = nullptr;
    __ONE_GPU_THREAD_END__
    data = tmp_data[INSTANCE_IDX_PER_BLOCK];
}

__host__ __device__ byte_array_t &byte_array_t::operator=(const byte_array_t &other) {
    __SHARED_MEMORY__ uint8_t *tmp_data[CGBN_IBP];
    // #ifdef __CUDA_ARCH__
    //     printf("byte_array_t::operator= %d this %p other %p\n", threadIdx.x, this, &other);
    //     printf("byte_array_t::operator= %d this size %d other size %d\n", threadIdx.x, size, other.size);
    //     printf("byte_array_t::operator= %d this data %p other data %p tmp_data before allocated %p\n", threadIdx.x,
    //     data,
    //            other.data, tmp_data);
    // #endif
    if (this != &other) {
        // if (size != other.size) {
        // __ONE_GPU_THREAD_BEGIN__
        // free();
        // tmp_data = (other.size > 0) ? new uint8_t[other.size] : nullptr;
        // __ONE_GPU_THREAD_END__
        // } else {
        // tmp_data = data; // cannot assign like this as data will be freed by others
        // }
        free();  // can do outside, all thread set to null

        if (other.size > 0) {
            // printf("other size !=0 %p \n", tmp_data);
            __ONE_GPU_THREAD_WOSYNC_BEGIN__
            tmp_data[INSTANCE_IDX_PER_BLOCK] = new uint8_t[other.size];
            __ONE_GPU_THREAD_END__

            // memcpy(tmp_data, other.data, other.size * sizeof(uint8_t));
            // parallel_copy
            uint32_t my_idx = THREAD_IDX_PER_INSTANCE;
            for (; my_idx < other.size; my_idx += CuEVM::cgbn_tpi) {
                tmp_data[INSTANCE_IDX_PER_BLOCK][my_idx] = other.data[my_idx];
            }
        } else
            tmp_data[INSTANCE_IDX_PER_BLOCK] = nullptr;

        data = tmp_data[INSTANCE_IDX_PER_BLOCK];
        size = other.size;
    }
    // __SYNC_THREADS__
    // #ifdef __CUDA_ARCH__
    //     printf("byte_array_t::operator= After %d this data %p other data %p\n", threadIdx.x, data, other.data);
    // #endif
    return *this;
}

__host__ __device__ int32_t byte_array_t::grow(uint32_t new_size, int32_t zero_padding) {
    // printf("byte_array_t::grow %d %d size %d zero_padding %d, new_size %d, data %p\n", THREADIDX,
    //        THREAD_IDX_PER_INSTANCE, size, zero_padding, new_size, data);
    if (new_size == size) return ERROR_SUCCESS;
    __SHARED_MEMORY__ uint8_t *new_data[CGBN_IBP];
    __ONE_GPU_THREAD_WOSYNC_BEGIN__
    new_data[INSTANCE_IDX_PER_BLOCK] = new uint8_t[new_size];
    __ONE_GPU_THREAD_END__
    // printf("byte_array_t::grow %d %d %d %d new_data_pointer %p\n", THREADIDX, THREAD_IDX_PER_INSTANCE, new_size,
    //        zero_padding, new_data[INSTANCE_IDX_PER_BLOCK]);
    if (zero_padding) {
        // parallel_copy
        uint32_t my_idx = THREAD_IDX_PER_INSTANCE;
        for (; my_idx < new_size; my_idx += CuEVM::cgbn_tpi) {
            new_data[INSTANCE_IDX_PER_BLOCK][my_idx] = 0;
        }
    }
    //  memset(new_data, 0, new_size * sizeof(uint8_t));
    if (size > 0) {
        // if (new_size > size) {
        // memcpy(new_data, data, min(new_size, size) * sizeof(uint8_t));
        // parallel_copy
        uint32_t my_idx = THREAD_IDX_PER_INSTANCE;
        for (; my_idx < min(new_size, size); my_idx += CuEVM::cgbn_tpi) {
            new_data[INSTANCE_IDX_PER_BLOCK][my_idx] = data[my_idx];
        }
        // if (zero_padding)
        //   memset(new_data + size, 0, new_size - size);
        // } else {
        //     memcpy(new_data, data, new_size * sizeof(uint8_t));
        // }
        __ONE_GPU_THREAD_WOSYNC_BEGIN__
        delete[] data;
        __ONE_GPU_THREAD_WOSYNC_END__
    }

    data = new_data[INSTANCE_IDX_PER_BLOCK];
    size = new_size;
    return ERROR_SUCCESS;
}

__host__ __device__ uint32_t byte_array_t::has_value(uint8_t value) const {
    __SHARED_MEMORY__ uint32_t error_code[CGBN_IBP];
    uint32_t index;
    error_code[INSTANCE_IDX_PER_BLOCK] = ERROR_VALUE_NOT_FOUND;
    __SYNC_THREADS__
#ifdef __CUDA_ARCH__
    uint32_t slot_size = size / CuEVM::cgbn_tpi;
    for (index = 0; index < slot_size; index++) {
        if (data[slot_size * threadIdx.x + index] == value) {
            error_code[INSTANCE_IDX_PER_BLOCK] = ERROR_SUCCESS;
        }
    }
    for (index = slot_size * CuEVM::cgbn_tpi; index < size; index++) {
        if (data[index] == value) {
            error_code[INSTANCE_IDX_PER_BLOCK] = ERROR_SUCCESS;
        }
    }
    __SYNC_THREADS__
#else
    for (index = 0; index < size; index++) {
        if (data[index] == value) {
            return ERROR_SUCCESS;
        }
    }
#endif
    return error_code[INSTANCE_IDX_PER_BLOCK];
}

__host__ __device__ void byte_array_t::print() const {
    __ONE_GPU_THREAD_WOSYNC_BEGIN__
    printf("size: %u\n", size);
    printf("data: ");
    for (uint32_t index = 0; index < size; index++) printf("%02x", data[index]);
    printf("\n");
    __ONE_GPU_THREAD_WOSYNC_END__
}

__host__ __device__ char *byte_array_t::to_hex() const {
    char *hex_string = new char[size * 2 + 3];  // 3 - 0x and \0
    hex_string[0] = '0';
    hex_string[1] = 'x';
    char *tmp_hex_string = (char *)hex_string + 2;
    uint8_t *tmp_data = data;
    for (uint32_t idx = 0; idx < size; idx++) {
        CuEVM::utils::hex_from_byte(tmp_hex_string, *(tmp_data++));
        tmp_hex_string += 2;
    }
    hex_string[size * 2 + 2] = 0;
    return hex_string;
}

__host__ cJSON *byte_array_t::to_json() const {
    cJSON *data_json = cJSON_CreateObject();
    cJSON_AddNumberToObject(data_json, "size", size);
    if (size > 0) {
        char *hex_string = to_hex();
        cJSON_AddStringToObject(data_json, "data", hex_string);
        delete[] hex_string;
    } else {
        cJSON_AddStringToObject(data_json, "data", "0x");
    }
    return data_json;
}

__host__ int32_t byte_array_t::from_hex_set_le(const char *clean_hex_string, int32_t length) {
    if ((length < 0) || ((size * 2) < length)) {
        return 1;
    }
    if (length > 0) {
        char *current_char;
        current_char = (char *)clean_hex_string;
        int32_t index;
        uint8_t *dst_ptr;
        dst_ptr = data;
        for (index = 0; index < ((length + 1) / 2) - 1; index++) {
            *(dst_ptr++) = CuEVM::utils::byte_from_two_hex_char(*(current_char), *(current_char + 1));
            current_char += 2;
        }
        if (length % 2 == 1) {
            *(dst_ptr++) = CuEVM::utils::byte_from_two_hex_char(*current_char++, '0');
        } else {
            *(dst_ptr++) = CuEVM::utils::byte_from_two_hex_char(*(current_char), *(current_char + 1));
            current_char += 2;
        }
    }
    return 0;
}

__host__ int32_t byte_array_t::from_hex_set_be(const char *clean_hex_string, int32_t length, PaddingDirection padding) {
    if ((length < 0) || ((size * 2) < length)) {
        return 1;
    }
    if (length > 0) {
        char *current_char;
        current_char = (char *)clean_hex_string;
        uint8_t *dst_ptr;
        if (padding == PaddingDirection::RIGHT_PADDING) {  // right padding
            dst_ptr = data + size - 1;
        } else if (padding == PaddingDirection::LEFT_PADDING) {  // left padding
            dst_ptr = data + (length + 1) / 2 - 1;
        } else {
            return 1;
        }

        if (length % 2 == 1) {
            *dst_ptr-- = CuEVM::utils::byte_from_two_hex_char('0', *current_char++);
        } else {
            *dst_ptr-- = CuEVM::utils::byte_from_two_hex_char(*(current_char), *(current_char + 1));
            current_char += 2;
        }
        while (*current_char != '\0') {
            *dst_ptr-- = CuEVM::utils::byte_from_two_hex_char(*(current_char), *(current_char + 1));
            current_char += 2;
        }
    }
    return 0;
}

__host__ int32_t byte_array_t::from_hex(const char *hex_string, int32_t endian, PaddingDirection padding,
                                        int32_t managed) {
    char *tmp_hex_char;
    tmp_hex_char = (char *)hex_string;
    int32_t length = CuEVM::utils::clean_hex_string(&tmp_hex_char);
    if (length < 0) {
        data = nullptr;
        return ERROR_INVALID_HEX_STRING;
    }
    uint32_t new_size = (size == 0) ? (length + 1) / 2 : size;
    if (size > 0) {
        if (managed) {
            free_managed();
        } else {
            free();
        }
    }
    size = new_size;
    if (size > 0) {
        if (managed) {
            CUDA_CHECK(hipMallocManaged((void **)&data, sizeof(uint8_t) * size));
            memset(data, 0, size * sizeof(uint8_t));
        } else {
            // data = (uint8_t*) std::calloc(size, sizeof(uint8_t));
            data = new uint8_t[size];
            memset(data, 0, size * sizeof(uint8_t));
        }
    } else
        data = nullptr;
    int32_t error_code = ERROR_SUCCESS;
    if (endian == LITTLE_ENDIAN) {
        error_code = this->from_hex_set_le(tmp_hex_char, length);
    } else {
        error_code = this->from_hex_set_be(tmp_hex_char, length, padding);
    }
    if (error_code != ERROR_SUCCESS) {
        if (managed) {
            CUDA_CHECK(hipFree(data));
        } else {
            delete[] data;
        }
        data = nullptr;
        size = 0;
    }
    return error_code;
}

__host__ __device__ int32_t byte_array_t::padded_copy_BE(const byte_array_t src) {
    uint32_t copy_size;
    int32_t size_diff;
    if (src.size == size) {
        size_diff = 0;
        copy_size = src.size;
    } else if (src.size < size) {
        size_diff = 1;
        copy_size = src.size;
    } else {
        size_diff = -1;
        copy_size = size;
    }
    __ONE_GPU_THREAD_BEGIN__
    memcpy(data, src.data, copy_size);
    memset(data + copy_size, 0, size - src.size);
    __ONE_GPU_THREAD_END__
    return size_diff;
}

__host__ __device__ uint8_t &byte_array_t::operator[](uint32_t index) { return data[index]; }

// STATIC FUNCTIONS

__host__ byte_array_t *byte_array_t::get_cpu(uint32_t count) {
    byte_array_t *cpu_instances = new byte_array_t[count];
    return cpu_instances;
}

__host__ void byte_array_t::cpu_free(byte_array_t *cpu_instances, uint32_t count) { delete[] cpu_instances; }

__host__ byte_array_t *byte_array_t::gpu_from_cpu(byte_array_t *cpu_instances, uint32_t count) {
    byte_array_t *gpu_instances, *tmp_cpu_instances;
    tmp_cpu_instances = new byte_array_t[count];
    for (uint32_t idx = 0; idx < count; idx++) {
        if (cpu_instances[idx].size > 0) {
            CUDA_CHECK(hipMalloc((void **)&tmp_cpu_instances[idx].data, sizeof(uint8_t) * cpu_instances[idx].size));
            CUDA_CHECK(hipMemcpy(tmp_cpu_instances[idx].data, cpu_instances[idx].data,
                                  sizeof(uint8_t) * cpu_instances[idx].size, hipMemcpyHostToDevice));
            tmp_cpu_instances[idx].size = cpu_instances[idx].size;
        }
    }
    CUDA_CHECK(hipMalloc((void **)&gpu_instances, sizeof(byte_array_t) * count));
    CUDA_CHECK(hipMemcpy(gpu_instances, tmp_cpu_instances, sizeof(byte_array_t) * count, hipMemcpyHostToDevice));
    for (uint32_t idx = 0; idx < count; idx++) {
        tmp_cpu_instances[idx].size = 0;
        tmp_cpu_instances[idx].data = nullptr;
    }
    delete[] tmp_cpu_instances;
    return gpu_instances;
}

__host__ void byte_array_t::gpu_free(byte_array_t *gpu_instances, uint32_t count) {
    byte_array_t *cpu_instances = new byte_array_t[count];
    CUDA_CHECK(hipMemcpy(cpu_instances, gpu_instances, sizeof(byte_array_t) * count, hipMemcpyDeviceToHost));
    for (uint32_t idx = 0; idx < count; idx++) {
        if (cpu_instances[idx].size > 0) {
            CUDA_CHECK(hipFree(cpu_instances[idx].data));
            cpu_instances[idx].data = nullptr;
            cpu_instances[idx].size = 0;
        }
    }
    delete[] cpu_instances;
    CUDA_CHECK(hipFree(gpu_instances));
}

__host__ byte_array_t *byte_array_t::cpu_from_gpu(byte_array_t *gpu_instances, uint32_t count) {
    byte_array_t *cpu_instances;
    cpu_instances = new byte_array_t[count];
    CUDA_CHECK(hipMemcpy(cpu_instances, gpu_instances, sizeof(byte_array_t) * count, hipMemcpyDeviceToHost));

    // 1. alocate the memory for gpu memory as memory which can be addressed by
    // the cpu
    byte_array_t *tmp_cpu_instances, *tmp_gpu_instances;
    tmp_cpu_instances = new byte_array_t[count];
    for (uint32_t idx = 0; idx < count; idx++) {
        if (cpu_instances[idx].size > 0) {
            CUDA_CHECK(hipMalloc((void **)&tmp_cpu_instances[idx].data, sizeof(uint8_t) * cpu_instances[idx].size));
            tmp_cpu_instances[idx].size = cpu_instances[idx].size;
        }
    }
    CUDA_CHECK(hipMalloc((void **)&tmp_gpu_instances, sizeof(byte_array_t) * count));
    CUDA_CHECK(hipMemcpy(tmp_gpu_instances, tmp_cpu_instances, sizeof(byte_array_t) * count, hipMemcpyHostToDevice));

    // 2. call the kernel to copy the memory between the gpu memories
    CUDA_CHECK(hipDeviceSynchronize());
    CuEVM::byte_array_t_transfer_kernel<<<count, 1>>>(tmp_gpu_instances, gpu_instances, count);
    CUDA_CHECK(hipDeviceSynchronize());
    hipFree(gpu_instances);
    gpu_instances = tmp_gpu_instances;
    tmp_gpu_instances = nullptr;

    // 3. copy the gpu memories back in the cpu memories
    CUDA_CHECK(hipMemcpy(cpu_instances, gpu_instances, sizeof(byte_array_t) * count, hipMemcpyDeviceToHost));
    for (uint32_t idx = 0; idx < count; idx++) {
        if (cpu_instances[idx].size > 0) {
            tmp_cpu_instances[idx].data = new uint8_t[cpu_instances[idx].size];
            CUDA_CHECK(hipMemcpy(tmp_cpu_instances[idx].data, cpu_instances[idx].data,
                                  sizeof(uint8_t) * cpu_instances[idx].size, hipMemcpyDeviceToHost));
            tmp_cpu_instances[idx].size = cpu_instances[idx].size;
        } else {
            tmp_cpu_instances[idx].data = nullptr;
            tmp_cpu_instances[idx].size = 0;
        }
    }
    // 4. free the temporary allocated memory
    byte_array_t::gpu_free(gpu_instances, count);
    for (uint32_t idx = 0; idx < count; idx++) {
        cpu_instances[idx].data = tmp_cpu_instances[idx].data;
        tmp_cpu_instances[idx].data = nullptr;
        tmp_cpu_instances[idx].size = 0;
    }
    delete[] tmp_cpu_instances;
    tmp_cpu_instances = nullptr;
    return cpu_instances;
}

__host__ __device__ void byte_array_t::transfer_memory(byte_array_t &dst, byte_array_t &src) {
    dst.size = src.size;
    if (src.size > 0) {
        memcpy(dst.data, src.data, src.size * sizeof(uint8_t));
    } else {
        dst.data = nullptr;
    }
    src.free();
}

// CPU-GPU
__global__ void byte_array_t_transfer_kernel(byte_array_t *dst_instances, byte_array_t *src_instances, uint32_t count) {
    uint32_t instance = blockIdx.x * blockDim.x + threadIdx.x;

    if (instance >= count) return;

    CuEVM::byte_array_t::transfer_memory(dst_instances[instance], src_instances[instance]);
}
}  // namespace CuEVM
