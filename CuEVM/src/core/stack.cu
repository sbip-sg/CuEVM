#include "hip/hip_runtime.h"
// CuEVM: CUDA Ethereum Virtual Machine implementation
// Copyright 2023 Stefan-Dan Ciocirlan (SBIP - Singapore Blockchain Innovation
// Programme) Author: Stefan-Dan Ciocirlan Date: 2023-11-30
// SPDX-License-Identifier: MIT

#include <CuEVM/core/stack.cuh>
#include <CuEVM/utils/error_codes.cuh>

namespace CuEVM::stack {

__host__ __device__ evm_stack_t::evm_stack_t() : stack_base(nullptr), stack_offset(0), capacity(0) {}

__host__ __device__ evm_stack_t::~evm_stack_t() { free(); }

__host__ __device__ evm_stack_t::evm_stack_t(const evm_stack_t &other) {
    // free();
    duplicate(other);
}

__host__ __device__ void evm_stack_t::free() {
    __ONE_GPU_THREAD_BEGIN__
    if (stack_base != nullptr) {
        delete[] stack_base;
    }
    __ONE_GPU_THREAD_END__
    clear();
}

__host__ __device__ void evm_stack_t::clear() {
    stack_offset = 0;
    capacity = 0;
    stack_base = nullptr;
}

__host__ __device__ evm_stack_t &evm_stack_t::operator=(const evm_stack_t &other) {
    if (this != &other) {
        free();
        duplicate(other);
    }
    return *this;
}

__host__ __device__ void evm_stack_t::duplicate(const evm_stack_t &other) {
    __SHARED_MEMORY__ evm_word_t *tmp_stack_base;
    __ONE_GPU_THREAD_BEGIN__
    tmp_stack_base = new evm_word_t[other.capacity];
    if (tmp_stack_base != nullptr) {
        memcpy(tmp_stack_base, other.stack_base, other.stack_offset * sizeof(evm_word_t));
    }
    __ONE_GPU_THREAD_END__
    stack_offset = other.stack_offset;
    capacity = other.stack_offset;
    stack_base = tmp_stack_base;
}

__host__ __device__ int32_t evm_stack_t::grow() {
    capacity = (capacity == 0) ? initial_capacity : capacity * 2;
    if (capacity > max_size) {
        return ERROR_STACK_OVERFLOW;
    }
    __SHARED_MEMORY__ evm_word_t *new_stack_base;
    __ONE_GPU_THREAD_BEGIN__
    new_stack_base = new evm_word_t[capacity];
    if (stack_base != nullptr && new_stack_base != nullptr) {
        memcpy(new_stack_base, stack_base, stack_offset * sizeof(evm_word_t));
        delete[] stack_base;
    }
    __ONE_GPU_THREAD_END__
    if (new_stack_base == nullptr) {
        return ERROR_MEMORY_ALLOCATION_FAILED;
    }
    stack_base = new_stack_base;
    return ERROR_SUCCESS;
}

__host__ __device__ uint32_t evm_stack_t::size() const { return stack_offset; }

__host__ __device__ evm_word_t *evm_stack_t::top() { return stack_base + stack_offset; }

__host__ __device__ int32_t evm_stack_t::push(ArithEnv &arith, const bn_t &value) {
    int32_t error_code = (size() >= capacity) ? grow() : ERROR_SUCCESS;
    if (error_code == ERROR_SUCCESS) {
        cgbn_store(arith.env, top(), value);
        stack_offset++;
    }

    return error_code;
}

__host__ __device__ int32_t evm_stack_t::pop(ArithEnv &arith, bn_t &y) {
    if (size() == 0) {
        // TODO: delete maybe?
        cgbn_set_ui32(arith.env, y, 0);
        return ERROR_STACK_UNDERFLOW;
    }
    stack_offset--;
    cgbn_load(arith.env, y, top());
    return ERROR_SUCCESS;
}

#ifdef __CUDA_ARCH__
__host__ __device__ int32_t evm_stack_t::pushx(ArithEnv &arith, uint8_t x, uint8_t *src_byte_data,
                                               uint8_t src_byte_size) {
    if (x > 32) {
        return ERROR_STACK_INVALID_SIZE;
    }
    // __ONE_GPU_THREAD_WOSYNC_BEGIN__
    // printf("pushx %d data size %d data to insert: \n", x, src_byte_size);
    // for (uint8_t idx = 0; idx < src_byte_size; idx++) {
    //     printf("%02x", src_byte_data[idx]);
    // }
    // printf("\n");
    // __ONE_GPU_THREAD_WOSYNC_END__

    int32_t error_code = (size() >= capacity) ? grow() : ERROR_SUCCESS;
    if (error_code == ERROR_SUCCESS) {
        int last_idx_from_left = 31 - min(x, src_byte_size);
        int my_idx = threadIdx.x % CuEVM::cgbn_tpi;
        // printf("pushx data inserted %d myidx %d firstidxleft %d\n", threadIdx.x, my_idx, last_idx_from_left);
        // if (my_idx < first_idx / 4) {
        //     top_->_limbs[my_idx] = 0;
        // } else {
        // each thead will insert 4 bytes/ hardcoded big endian for now
        int byte_start = (my_idx + 1) * 4 - 1;
        uint32_t limb_value = 0;
        if (byte_start > last_idx_from_left) {
            limb_value |= src_byte_data[src_byte_size - 1 - (31 - byte_start)];  //<< 24;
        }
        byte_start--;
        if (byte_start > last_idx_from_left) {
            limb_value |= src_byte_data[src_byte_size - 1 - (31 - byte_start)] << 8;  // << 16;
        }
        byte_start--;
        if (byte_start > last_idx_from_left) {
            limb_value |= src_byte_data[src_byte_size - 1 - (31 - byte_start)] << 16;  // << 8;
        }
        byte_start--;
        if (byte_start > last_idx_from_left) {
            // printf("bytestart %d src data idx %d thread idx %d, srcbyte %02x \n", byte_start, 31 - byte_start,
            //        threadIdx.x, src_byte_data[31 - byte_start]);
            limb_value |= src_byte_data[src_byte_size - 1 - (31 - byte_start)] << 24;
        }

        top()->_limbs[CuEVM::cgbn_tpi - my_idx - 1] = limb_value;
    }

    // __SYNC_THREADS__  // do we need to sync here?
    //     printf("pushx data inserted %d\n", threadIdx.x);
    // top()->print();

    stack_offset++;
    return error_code;
}

#else
__host__ __device__ int32_t evm_stack_t::pushx(ArithEnv &arith, uint8_t x, uint8_t *src_byte_data,
                                               uint8_t src_byte_size) {
    // TODO:: for sure is something more efficient here
    if (x > 32) {
        return ERROR_STACK_INVALID_SIZE;
    }
    bn_t r;
    cgbn_set_ui32(arith.env, r, 0);
    for (uint8_t idx = (x - src_byte_size); idx < x; idx++) {
        cgbn_insert_bits_ui32(arith.env, r, r, idx * 8, 8, src_byte_data[x - 1 - idx]);
    }

    return push(arith, r);
}
#endif

__host__ __device__ int32_t evm_stack_t::get_index(ArithEnv &arith, uint32_t index, bn_t &y) {
    if (index > size()) {
        return ERROR_STACK_INVALID_INDEX;
    }
    cgbn_load(arith.env, y, stack_base + size() - index);
    return ERROR_SUCCESS;
}

__host__ __device__ int32_t evm_stack_t::dupx(ArithEnv &arith, uint32_t x) {
    bn_t r;
    int32_t error_code = ((x > 16) || (x < 1)) ? ERROR_STACK_INVALID_SIZE : get_index(arith, x, r);
    return error_code | push(arith, r);
}

__host__ __device__ int32_t evm_stack_t::swapx(ArithEnv &arith, uint32_t x) {
    bn_t a, b;
    int32_t error_code =
        ((x > 16) || (x < 1)) ? ERROR_STACK_INVALID_SIZE : (get_index(arith, 1, a) | get_index(arith, x + 1, b));
    if (error_code == ERROR_SUCCESS) {
        cgbn_store(arith.env, stack_base + size() - x - 1, a);
        cgbn_store(arith.env, stack_base + size() - 1, b);
    }
    return error_code;
}

__host__ __device__ void evm_stack_t::print() {
    __ONE_GPU_THREAD_WOSYNC_BEGIN__
    printf("Stack size: %d, data:\n", size());
    for (uint32_t idx = 0; idx < size(); idx++) {
        stack_base[idx].print();
    }
    __ONE_GPU_THREAD_WOSYNC_END__
}

__host__ cJSON *evm_stack_t::to_json() {
    cJSON *json = cJSON_CreateObject();
    char *hex_string_ptr = new char[CuEVM::word_size * 2 + 3];
    cJSON *stack = cJSON_CreateArray();
    for (uint32_t idx = 0; idx < size(); idx++) {
        stack_base[idx].to_hex(hex_string_ptr);
        cJSON_AddItemToArray(stack, cJSON_CreateString(hex_string_ptr));
    }
    cJSON_AddItemToObject(json, "data", stack);
    delete[] hex_string_ptr;
    return json;
}
__host__ evm_stack_t *evm_stack_t::get_cpu(uint32_t count) {
    evm_stack_t *instances = new evm_stack_t[count];
    return instances;
}
__host__ void evm_stack_t::cpu_free(evm_stack_t *instances, uint32_t count) { delete[] instances; }
__host__ evm_stack_t *evm_stack_t::gpu_from_cpu(evm_stack_t *cpu_instances, uint32_t count) {
    evm_stack_t *gpu_instances, *tmp_gpu_instances;
    tmp_gpu_instances = new evm_stack_t[count];
    for (uint32_t idx = 0; idx < count; idx++) {
        if (cpu_instances[idx].stack_base != nullptr) {
            CUDA_CHECK(
                hipMalloc(&tmp_gpu_instances[idx].stack_base, cpu_instances[idx].capacity * sizeof(evm_word_t)));
            CUDA_CHECK(hipMemcpy(tmp_gpu_instances[idx].stack_base, cpu_instances[idx].stack_base,
                                  cpu_instances[idx].stack_offset * sizeof(evm_word_t), hipMemcpyHostToDevice));
        } else {
            tmp_gpu_instances[idx].stack_base = nullptr;
        }
        tmp_gpu_instances[idx].stack_offset = cpu_instances[idx].stack_offset;
        tmp_gpu_instances[idx].capacity = cpu_instances[idx].capacity;
    }
    CUDA_CHECK(hipMalloc(&gpu_instances, count * sizeof(evm_stack_t)));
    CUDA_CHECK(hipMemcpy(gpu_instances, tmp_gpu_instances, count * sizeof(evm_stack_t), hipMemcpyHostToDevice));
    for (uint32_t idx = 0; idx < count; idx++) {
        tmp_gpu_instances[idx].clear();
    }
    delete[] tmp_gpu_instances;
    return gpu_instances;
}
__host__ void evm_stack_t::gpu_free(evm_stack_t *gpu_instances, uint32_t count) {
    evm_stack_t *tmp_gpu_instances = new evm_stack_t[count];
    CUDA_CHECK(hipMemcpy(tmp_gpu_instances, gpu_instances, count * sizeof(evm_stack_t), hipMemcpyDeviceToHost));
    for (uint32_t idx = 0; idx < count; idx++) {
        if (tmp_gpu_instances[idx].stack_base != nullptr && tmp_gpu_instances[idx].capacity > 0 &&
            tmp_gpu_instances[idx].stack_offset > 0) {
            CUDA_CHECK(hipFree(tmp_gpu_instances[idx].stack_base));
        }
        tmp_gpu_instances[idx].clear();
    }
    delete[] tmp_gpu_instances;
    CUDA_CHECK(hipFree(gpu_instances));
}

__host__ evm_stack_t *evm_stack_t::cpu_from_gpu(evm_stack_t *gpu_instances, uint32_t count) {
    evm_stack_t *cpu_instances = new evm_stack_t[count];
    evm_stack_t *tmp_gpu_instances = new evm_stack_t[count];
    evm_stack_t *tmp_cpu_instances = new evm_stack_t[count];
    CUDA_CHECK(hipMemcpy(cpu_instances, gpu_instances, count * sizeof(evm_stack_t), hipMemcpyDeviceToHost));
    for (uint32_t idx = 0; idx < count; idx++) {
        if (cpu_instances[idx].stack_offset > 0) {
            CUDA_CHECK(
                hipMalloc(&tmp_cpu_instances[idx].stack_base, cpu_instances[idx].stack_offset * sizeof(evm_word_t)));
        } else {
            tmp_cpu_instances[idx].stack_base = nullptr;
        }
        tmp_cpu_instances[idx].stack_offset = cpu_instances[idx].stack_offset;
        tmp_cpu_instances[idx].capacity = cpu_instances[idx].stack_offset;
    }
    CUDA_CHECK(hipMalloc(&tmp_gpu_instances, count * sizeof(evm_stack_t)));
    CUDA_CHECK(hipMemcpy(tmp_gpu_instances, tmp_cpu_instances, count * sizeof(evm_stack_t), hipMemcpyHostToDevice));
    for (uint32_t idx = 0; idx < count; idx++) {
        tmp_cpu_instances[idx].clear();
        cpu_instances[idx].clear();
    }
    transfer_kernel_evm_stack_t<<<count, 1>>>(tmp_gpu_instances, gpu_instances, count);
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipFree(gpu_instances));

    CUDA_CHECK(hipMemcpy(tmp_cpu_instances, tmp_gpu_instances, sizeof(evm_stack_t) * count, hipMemcpyDeviceToHost));

    for (uint32_t idx = 0; idx < count; idx++) {
        if (tmp_cpu_instances[idx].stack_offset > 0) {
            cpu_instances[idx].stack_base = new evm_word_t[tmp_cpu_instances[idx].stack_offset];
            CUDA_CHECK(hipMemcpy(cpu_instances[idx].stack_base, tmp_cpu_instances[idx].stack_base,
                                  sizeof(evm_word_t) * tmp_cpu_instances[idx].stack_offset, hipMemcpyDeviceToHost));
        } else {
            cpu_instances[idx].stack_base = NULL;
        }
        cpu_instances[idx].stack_offset = tmp_cpu_instances[idx].stack_offset;
        cpu_instances[idx].capacity = tmp_cpu_instances[idx].stack_offset;
    }
    for (uint32_t idx = 0; idx < count; idx++) {
        tmp_cpu_instances[idx].clear();
    }
    delete[] tmp_cpu_instances;
    evm_stack_t::gpu_free(tmp_gpu_instances, count);
    return cpu_instances;
}

__global__ void transfer_kernel_evm_stack_t(evm_stack_t *dst, evm_stack_t *src, uint32_t count) {
    uint32_t instance = blockIdx.x * blockDim.x + threadIdx.x;
    if (instance >= count) {
        return;
    }
    dst[instance].stack_offset = src[instance].stack_offset;
    dst[instance].capacity = src[instance].stack_offset;
    memcpy(dst[instance].stack_base, src[instance].stack_base, src[instance].stack_offset * sizeof(evm_word_t));
    delete[] src[instance].stack_base;
    src[instance].clear();
}

}  // namespace CuEVM::stack