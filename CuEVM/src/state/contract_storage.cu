#include "hip/hip_runtime.h"
// CuEVM: CUDA Ethereum Virtual Machine implementation
// Copyright 2023 Stefan-Dan Ciocirlan (SBIP - Singapore Blockchain Innovation
// Programme) Author: Stefan-Dan Ciocirlan Date: 2024-09-15
// SPDX-License-Identifier: MIT

#include <CuEVM/state/contract_storage.cuh>
#include <CuEVM/utils/error_codes.cuh>

namespace CuEVM {

__host__ __device__ contract_storage_t::~contract_storage_t() { free(); }

__host__ __device__ void contract_storage_t::free() {
    __ONE_GPU_THREAD_WOSYNC_BEGIN__
    if ((storage != nullptr) && (capacity > 0)) {
        delete[] storage;
    }
    __ONE_GPU_THREAD_WOSYNC_END__
    clear();
}

__host__ void contract_storage_t::free_managed() {
    if (storage != nullptr && capacity > 0) {
        CUDA_CHECK(hipFree(storage));
    }
    clear();
}

__host__ __device__ void contract_storage_t::clear() {
    storage = nullptr;
    size = 0;
    capacity = 0;
}

__host__ __device__ contract_storage_t &contract_storage_t::operator=(const contract_storage_t &other) {
    __SHARED_MEMORY__ storage_element_t *tmp_storage[CGBN_IBP];
    // __ONE_GPU_THREAD_WOSYNC_BEGIN__
    if (this == &other) {
        return *this;
    }
    if (capacity != other.capacity) {
        free();
        size = other.size;
        capacity = other.capacity;
        // #ifdef __CUDA_ARCH__
        //         printf("contract_storage_t::operator= idx %d size %d capacity %d\n", threadIdx.x, size, capacity);
        // #endif
        __ONE_GPU_THREAD_BEGIN__
        if (capacity > 0) {
            tmp_storage[INSTANCE_IDX_PER_BLOCK] = new storage_element_t[capacity];
        }
        __ONE_GPU_THREAD_END__
        storage = tmp_storage[INSTANCE_IDX_PER_BLOCK];
    }
    __ONE_GPU_THREAD_BEGIN__
    if (other.size > 0) memcpy(storage, other.storage, other.size * sizeof(storage_element_t));
    __ONE_GPU_THREAD_END__
    // __ONE_GPU_THREAD_WOSYNC_END__
    return *this;
}

__host__ __device__ int32_t contract_storage_t::get_value(ArithEnv &arith, const bn_t &key, bn_t &value) const {
    uint32_t idx = 0;
    // #ifdef __CUDA_ARCH__
    //     printf("contract_storage_t::get_value idx %d size %d capacity %d storage %p\n", threadIdx.x, size, capacity,
    //            storage);
    // #endif
    for (idx = 0; idx < size; idx++) {
        if (storage[idx].has_key(arith, key)) {
            storage[idx].get_value(arith, value);
            return ERROR_SUCCESS;
        }
    }
    return ERROR_STORAGE_KEY_NOT_FOUND;
}

__host__ __device__ int32_t contract_storage_t::set_value(ArithEnv &arith, const bn_t &key, const bn_t &value) {
    uint32_t idx;
    __SYNC_THREADS__  // ? why is this needed?
                      // todo : remove the loop
        for (idx = 0; idx < size; idx++) {
        if (storage[idx].has_key(arith, key)) {
            storage[idx].set_value(arith, value);
            return ERROR_SUCCESS;
        }
    }

    __SHARED_MEMORY__ storage_element_t *new_storage[CGBN_IBP];
    // #ifdef __CUDA_ARCH__
    //     printf("contract_storage_t::set_value before allocateidx %d size %d capacity %d  storage %p\n", threadIdx.x,
    //     size,
    //            capacity, storage);
    // #endif
    if (size >= capacity) {
        if (capacity == 0) {
            capacity = CuEVM::initial_storage_capacity;
        } else {
            capacity *= 2;
        }
        __ONE_GPU_THREAD_WOSYNC_BEGIN__
        new_storage[INSTANCE_IDX_PER_BLOCK] = new storage_element_t[capacity];
        // printf("allocate new storage %p, capacity %d\n", new_storage, capacity);
        if (size > 0) {
            memcpy(new_storage[INSTANCE_IDX_PER_BLOCK], storage, size * sizeof(storage_element_t));
        }
        delete[] storage;
        __ONE_GPU_THREAD_END__
        storage = new_storage[INSTANCE_IDX_PER_BLOCK];
        // printf("set storage size %d capacity %d  storage %p, new_storage %p\n", size, capacity, storage,
        // new_storage);
    }
    // #ifdef __CUDA_ARCH__
    //     printf("contract_storage_t::set_value idx %d size %d capacity %d, storage %p\n", threadIdx.x, size, capacity,
    //     storage); printf("contract_storage_t::set_value idx %d storage 0 key %d val %d\n", threadIdx.x,
    //     storage[0].key._limbs[0], storage[0].value._limbs[0]);
    // #endif
    // #ifdef __CUDA_ARCH__
    //     printf("contract_storage_t::set_value idx %d size %d capacity %d, storage %p\n", threadIdx.x, size, capacity,
    //            storage);
    // #endif
    storage[size].set_key(arith, key);
    storage[size].set_value(arith, value);
    size++;
    return ERROR_SUCCESS;
}

__host__ __device__ void contract_storage_t::update(ArithEnv &arith, const contract_storage_t &other) {
    bn_t key, value;
    for (uint32_t idx = 0; idx < other.size; idx++) {
        cgbn_load(arith.env, key, (cgbn_evm_word_t_ptr)&other.storage[idx].key);
        cgbn_load(arith.env, value, (cgbn_evm_word_t_ptr)&other.storage[idx].value);
        set_value(arith, key, value);
    }
}

__host__ int32_t contract_storage_t::from_json(const cJSON *contract_storage_json, int32_t managed) {
    if (cJSON_IsNull(contract_storage_json) || cJSON_IsInvalid(contract_storage_json) /*||
                                               (!cJSON_IsArray(contract_storage_json))*/
    ) {
        return ERROR_INVALID_JSON;
    }
    size = cJSON_GetArraySize(contract_storage_json);
    if (size == 0) {
        capacity = 0;
        storage = nullptr;
        return ERROR_SUCCESS;
    }
    capacity = CuEVM::initial_storage_capacity / 2;
    do {
        capacity *= 2;
    } while (capacity < size);
    if (managed) {
        CUDA_CHECK(hipMallocManaged(&storage, capacity * sizeof(storage_element_t)));
    } else {
        storage = new storage_element_t[capacity];
    }
    cJSON *element_json = nullptr;
    uint32_t idx = 0;
    cJSON_ArrayForEach(element_json, contract_storage_json) {
        storage[idx].from_json(element_json);
        idx++;
    }
    return 0;
}

__host__ cJSON *contract_storage_t::to_json(int32_t pretty) const {
    cJSON *contract_storage_json = cJSON_CreateObject();
    if (size == 0) {
        return contract_storage_json;
    }
    uint32_t idx = 0;
    char *key_string_ptr = new char[CuEVM::word_size * 2 + 3];
    char *value_string_ptr = new char[CuEVM::word_size * 2 + 3];
    for (idx = 0; idx < size; idx++) {
        storage[idx].add_to_json(contract_storage_json, key_string_ptr, value_string_ptr, pretty);
    }
    delete[] key_string_ptr;
    delete[] value_string_ptr;
    return contract_storage_json;
}

__host__ __device__ void contract_storage_t::print() const {
    __ONE_GPU_THREAD_WOSYNC_BEGIN__
    printf("Storage size: %u\n", size);
    for (uint32_t idx = 0; idx < size; idx++) {
        printf("Element %u:\n", idx);
        storage[idx].print();
    }
    __ONE_GPU_THREAD_WOSYNC_END__
}

__host__ int32_t contract_storage_t::has_key(const evm_word_t &key, uint32_t &index) const {
    for (index = 0; index < size; index++) {
        if (storage[index].has_key(key)) {
            return ERROR_SUCCESS;
        }
    }
    return ERROR_STORAGE_KEY_NOT_FOUND;
}

__host__ cJSON *contract_storage_t::merge_json(const contract_storage_t &storage1, const contract_storage_t &storage2,
                                               const int32_t pretty) {
    cJSON *storage_json = cJSON_CreateObject();
    uint8_t *written = new uint8_t[storage2.size];
    memset(written, 0, storage2.size);
    char *key_string_ptr = new char[CuEVM::word_size * 2 + 3];
    char *value_string_ptr = new char[CuEVM::word_size * 2 + 3];
    for (uint32_t idx = 0; idx < storage1.size; idx++) {
        uint32_t jdx;
        if (storage2.has_key(storage1.storage[idx].key, jdx) == ERROR_SUCCESS) {
            storage2.storage[jdx].key.to_hex(key_string_ptr, pretty);
            storage2.storage[jdx].value.to_hex(value_string_ptr, pretty);
            written[jdx] = 1;
        } else {
            storage1.storage[idx].key.to_hex(key_string_ptr, pretty);
            storage1.storage[idx].value.to_hex(value_string_ptr, pretty);
        }

        // if value is different than 0
        if (value_string_ptr[2] != '0' || value_string_ptr[3] != '\0') {
            cJSON_AddStringToObject(storage_json, key_string_ptr, value_string_ptr);
        }
    }

    for (uint32_t jdx = 0; jdx < storage2.size; jdx++) {
        if (written[jdx] == 0) {
            storage2.storage[jdx].key.to_hex(key_string_ptr, pretty);
            storage2.storage[jdx].value.to_hex(value_string_ptr, pretty);
            if (value_string_ptr[2] != '0' || value_string_ptr[3] != '\0') {
                cJSON_AddStringToObject(storage_json, key_string_ptr, value_string_ptr);
            }
        }
    }
    delete[] written;
    delete[] key_string_ptr;
    delete[] value_string_ptr;
    return storage_json;
}

// __host__ __device__ void contract_storage_t::transfer_memory(contract_storage_t &dst, contract_storage_t &src) {
//     if ((src.size > 0) && (src.storage != nullptr) && (src.capacity > 0)) {
//         memcpy(dst.storage, src.storage, src.size * sizeof(storage_element_t));
//         dst.size = src.size;
//         dst.capacity = src.size;
//     } else {
//         // TODO: check if this is necessary
//         dst.size = 0;
//     }
//     src.free();
// }

__host__ contract_storage_t *contract_storage_t::get_cpu(uint32_t count) { return new contract_storage_t[count]; }

__host__ void contract_storage_t::cpu_free(contract_storage_t *contract_storages, uint32_t count) {
    delete[] contract_storages;
}

__host__ contract_storage_t *contract_storage_t::get_gpu_from_cpu(contract_storage_t *cpu_contract_storages,
                                                                  uint32_t count) {
    contract_storage_t *gpu_contract_storages, *tmp_contract_storages;
    tmp_contract_storages = new contract_storage_t[count];
    for (uint32_t idx = 0; idx < count; idx++) {
        if (cpu_contract_storages[idx].size > 0) {
            CUDA_CHECK(hipMalloc(&tmp_contract_storages[idx].storage,
                                  cpu_contract_storages[idx].capacity * sizeof(storage_element_t)));
            CUDA_CHECK(hipMemcpy(tmp_contract_storages[idx].storage, cpu_contract_storages[idx].storage,
                                  cpu_contract_storages[idx].size * sizeof(storage_element_t), hipMemcpyHostToDevice));
            tmp_contract_storages[idx].size = cpu_contract_storages[idx].size;
            tmp_contract_storages[idx].capacity = cpu_contract_storages[idx].capacity;
        } else {
            tmp_contract_storages[idx].clear();
        }
    }
    CUDA_CHECK(hipMalloc(&gpu_contract_storages, count * sizeof(contract_storage_t)));
    CUDA_CHECK(hipMemcpy(gpu_contract_storages, tmp_contract_storages, count * sizeof(contract_storage_t),
                          hipMemcpyHostToDevice));
    for (uint32_t idx = 0; idx < count; idx++) {
        tmp_contract_storages[idx].clear();
    }
    delete[] tmp_contract_storages;
    return gpu_contract_storages;
}

__host__ void contract_storage_t::gpu_free(contract_storage_t *gpu_contract_storages, uint32_t count) {
    contract_storage_t *tmp_contract_storages;
    tmp_contract_storages = new contract_storage_t[count];
    CUDA_CHECK(hipMemcpy(tmp_contract_storages, gpu_contract_storages, count * sizeof(contract_storage_t),
                          hipMemcpyDeviceToHost));
    for (uint32_t idx = 0; idx < count; idx++) {
        if (tmp_contract_storages[idx].capacity > 0) {
            CUDA_CHECK(hipFree(tmp_contract_storages[idx].storage));
        }
        tmp_contract_storages[idx].clear();
    }
    delete[] tmp_contract_storages;
    CUDA_CHECK(hipFree(gpu_contract_storages));
}

__host__ contract_storage_t *contract_storage_t::get_cpu_from_gpu(contract_storage_t *gpu_contract_storages,
                                                                  uint32_t count) {
    contract_storage_t *cpu_contract_storages, *tmp_contract_storages, *tmp_gpu_contract_storages;
    tmp_contract_storages = new contract_storage_t[count];
    cpu_contract_storages = new contract_storage_t[count];
    CUDA_CHECK(hipMemcpy(cpu_contract_storages, gpu_contract_storages, count * sizeof(contract_storage_t),
                          hipMemcpyDeviceToHost));
    for (uint32_t idx = 0; idx < count; idx++) {
        if (cpu_contract_storages[idx].size > 0) {
            CUDA_CHECK(hipMalloc(&tmp_contract_storages[idx].storage,
                                  cpu_contract_storages[idx].size * sizeof(storage_element_t)));
            tmp_contract_storages[idx].size = cpu_contract_storages[idx].size;
            tmp_contract_storages[idx].capacity = cpu_contract_storages[idx].size;
        } else {
            tmp_contract_storages[idx].clear();
        }
    }
    CUDA_CHECK(hipMalloc(&tmp_gpu_contract_storages, count * sizeof(contract_storage_t)));
    CUDA_CHECK(hipMemcpy(tmp_gpu_contract_storages, tmp_contract_storages, count * sizeof(contract_storage_t),
                          hipMemcpyHostToDevice));
    contract_storage_t_transfer_kernel<<<count, 1>>>(tmp_gpu_contract_storages, gpu_contract_storages, count);
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipFree(gpu_contract_storages));
    CUDA_CHECK(hipMemcpy(tmp_contract_storages, tmp_gpu_contract_storages, count * sizeof(contract_storage_t),
                          hipMemcpyDeviceToHost));
    for (uint32_t idx = 0; idx < count; idx++) {
        if (tmp_contract_storages[idx].capacity > 0) {
            cpu_contract_storages[idx].storage = new storage_element_t[tmp_contract_storages[idx].capacity];
            CUDA_CHECK(hipMemcpy(cpu_contract_storages[idx].storage, tmp_contract_storages[idx].storage,
                                  tmp_contract_storages[idx].size * sizeof(storage_element_t), hipMemcpyDeviceToHost));
            cpu_contract_storages[idx].size = tmp_contract_storages[idx].size;
            cpu_contract_storages[idx].capacity = tmp_contract_storages[idx].capacity;
        } else {
            cpu_contract_storages[idx].clear();
        }
        tmp_contract_storages[idx].clear();
    }
    delete[] tmp_contract_storages;
    contract_storage_t::gpu_free(tmp_gpu_contract_storages, count);
    return cpu_contract_storages;
}

__global__ void contract_storage_t_transfer_kernel(contract_storage_t *dst_instances, contract_storage_t *src_instances,
                                                   uint32_t instance_count) {
    uint32_t instance = blockIdx.x * blockDim.x + threadIdx.x;
    if (instance < instance_count) {
        // TODO Fix this later
        // contract_storage_t::transfer_memory(dst_instances[instance], src_instances[instance]);
    }
}

}  // namespace CuEVM