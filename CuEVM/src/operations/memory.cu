// CuEVM: CUDA Ethereum Virtual Machine implementation
// Copyright 2023 Stefan-Dan Ciocirlan (SBIP - Singapore Blockchain Innovation Programme)
// Author: Stefan-Dan Ciocirlan
// Data: 2023-11-30
// SPDX-License-Identifier: MIT

#include <CuEVM/gas_cost.cuh>
#include <CuEVM/operations/memory.cuh>
#include <CuEVM/utils/error_codes.cuh>

namespace CuEVM::operations {
__host__ __device__ int32_t MLOAD(ArithEnv &arith, const bn_t &gas_limit, bn_t &gas_used, CuEVM::evm_stack_t &stack,
                                  CuEVM::evm_memory_t &memory) {
    cgbn_add_ui32(arith.env, gas_used, gas_used, GAS_VERY_LOW);
    int32_t error_code = CuEVM::gas_cost::has_gas(arith, gas_limit, gas_used);

    bn_t memory_offset, length;
    error_code |= stack.pop(arith, memory_offset);
    cgbn_set_ui32(arith.env, length, CuEVM::word_size);

    // get the memory expansion gas cost
    bn_t memory_expansion_cost;
    error_code |=
        CuEVM::gas_cost::memory_grow_cost(arith, memory, memory_offset, length, memory_expansion_cost, gas_used);

    error_code |= CuEVM::gas_cost::has_gas(arith, gas_limit, gas_used);
    if (error_code == ERROR_SUCCESS) {
        memory.increase_memory_cost(arith, memory_expansion_cost);
        CuEVM::byte_array_t data;
        error_code |= memory.get(arith, memory_offset, length, data);

        bn_t value;
        error_code |= cgbn_set_byte_array_t(arith.env, value, data);
        error_code |= stack.push(arith, value);
    }
    return error_code;
}

__host__ __device__ int32_t MSTORE(ArithEnv &arith, const bn_t &gas_limit, bn_t &gas_used, CuEVM::evm_stack_t &stack,
                                   CuEVM::evm_memory_t &memory) {
    cgbn_add_ui32(arith.env, gas_used, gas_used, GAS_VERY_LOW);
    int32_t error_code = CuEVM::gas_cost::has_gas(arith, gas_limit, gas_used);

    bn_t memory_offset;
    error_code |= stack.pop(arith, memory_offset);
    bn_t value;
    error_code |= stack.pop(arith, value);

    bn_t length;
    cgbn_set_ui32(arith.env, length, CuEVM::word_size);

    // get the memory expansion gas cost
    bn_t memory_expansion_cost;
    error_code |=
        CuEVM::gas_cost::memory_grow_cost(arith, memory, memory_offset, length, memory_expansion_cost, gas_used);

    error_code |= CuEVM::gas_cost::has_gas(arith, gas_limit, gas_used);

    if (error_code == ERROR_SUCCESS) {
        memory.increase_memory_cost(arith, memory_expansion_cost);
        CuEVM::byte_array_t value_bytes(CuEVM::word_size);
        __SHARED_MEMORY__ evm_word_t value_word;
        cgbn_store(arith.env, (cgbn_evm_word_t_ptr)&value_word, value);

        value_word.to_byte_array_t(value_bytes);

        error_code |= memory.set(arith, value_bytes, memory_offset, length);
    }
    return error_code;
}

__host__ __device__ int32_t MSTORE8(ArithEnv &arith, const bn_t &gas_limit, bn_t &gas_used, CuEVM::evm_stack_t &stack,
                                    CuEVM::evm_memory_t &memory) {
    cgbn_add_ui32(arith.env, gas_used, gas_used, GAS_VERY_LOW);
    int32_t error_code = CuEVM::gas_cost::has_gas(arith, gas_limit, gas_used);
    // #ifdef __CUDA_ARCH__
    //     printf("MSTORE8 %d\n", threadIdx.x);
    // #endif
    bn_t memory_offset;
    error_code |= stack.pop(arith, memory_offset);
    bn_t value;
    error_code |= stack.pop(arith, value);
    bn_t length;
    cgbn_set_ui32(arith.env, length, 1);
    // get the memory expansion gas cost
    bn_t memory_expansion_cost;
    error_code |=
        CuEVM::gas_cost::memory_grow_cost(arith, memory, memory_offset, length, memory_expansion_cost, gas_used);

    error_code |= CuEVM::gas_cost::has_gas(arith, gas_limit, gas_used);

    if (error_code == ERROR_SUCCESS) {
        memory.increase_memory_cost(arith, memory_expansion_cost);
        CuEVM::byte_array_t value_bytes(CuEVM::word_size);
        // TODO: bnt directly to byte array
        __SHARED_MEMORY__ evm_word_t value_word;
        cgbn_store(arith.env, (cgbn_evm_word_t_ptr)&value_word, value);
        value_word.to_byte_array_t(value_bytes);

        CuEVM::byte_array_t value_byte(value_bytes.data + CuEVM::word_size - 1, 1);

        error_code |= memory.set(arith, value_byte, memory_offset, length);
    }
    return error_code;
}

__host__ __device__ int32_t MSIZE(ArithEnv &arith, const bn_t &gas_limit, bn_t &gas_used, CuEVM::evm_stack_t &stack,
                                  const CuEVM::evm_memory_t &memory) {
    cgbn_add_ui32(arith.env, gas_used, gas_used, GAS_BASE);
    int32_t error_code = CuEVM::gas_cost::has_gas(arith, gas_limit, gas_used);
    if (error_code == ERROR_SUCCESS) {
        bn_t size;
        cgbn_set_ui32(arith.env, size, memory.get_size());

        error_code |= stack.push(arith, size);
    }
    return error_code;
}
}  // namespace CuEVM::operations
